#include "hip/hip_runtime.h"
﻿#include "cuda_kernal.h"
#include ""
#include "math_functions.h"
#include "hip/hip_vector_types.h"

// Error checking macro
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            std::cerr << "CUDA error in " << __FILE__ << "(" << __LINE__ << "): " \
                      << hipGetErrorString(err) << std::endl;              \
            std::exit(EXIT_FAILURE);                                        \
        }                                                                   \
    } while (0)


Body* d_Bodies;
Body_Info* d_Bodies_ext;

inline float flerp(float min, float max, float t) {
    return min * (1 - t) + max * (t);
}
inline float frandom(float min, float max) {
    float frand = (float)rand() / (float)RAND_MAX;
    return flerp(min, max, frand);
}

__device__ uint8_t* d_render_texture;          //device texture ptr
__constant__ int2 d_texture_size;           //device texture size in pixels
__constant__ int d_channels;                //device channels
const int MAX_CHANNELS = 4;

uint8_t* h_render_texture;                 //texture ptr
int texture_mem_size;                   //texture size in bytes
int2 texture_size;                      //texture size in pixels

__device__ int2 get_pixel_coord() {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    return make_int2(x, y);
}

__device__ void set_pixel(uint8_t* texture, float4 color, int2 coord) {
    coord.y -= d_texture_size.y;
    coord.y *= -1;

    int ptr_loc = (coord.x + coord.y * d_texture_size.x) * MAX_CHANNELS;
    
    int red_offset = ptr_loc + min(0, d_channels - 1);
    int green_offset = ptr_loc + min(1, d_channels - 1);
    int blue_offset = ptr_loc + min(2, d_channels - 1);
    int alpha_offset = ptr_loc + min(3, d_channels - 1);
    color.x = min(color.x, 1.f);
    color.y = min(color.y, 1.f);
    color.z = min(color.z, 1.f);
    color.w = min(color.w, 1.f);
    *(texture + alpha_offset) = (uint8_t)(color.w * 255);
    *(texture + blue_offset) = (uint8_t)(color.z * 255);
    *(texture + green_offset) = (uint8_t)(color.y * 255);
    *(texture + red_offset) = (uint8_t)(color.x * 255);
}

__device__ float4 get_pixel(uint8_t* texture, int2 coord) {
    coord.y -= d_texture_size.y;
    coord.y *= -1;

    int ptr_loc = (coord.x + coord.y * d_texture_size.x) * MAX_CHANNELS;

    int red_offset = ptr_loc + min(0, d_channels - 1);
    int green_offset = ptr_loc + min(1, d_channels - 1);
    int blue_offset = ptr_loc + min(2, d_channels - 1);
    int alpha_offset = ptr_loc + min(3, d_channels - 1);
    float4 color = make_float4(0, 0, 0, 0);

    if (d_channels >= 4)
        color.w = (*(texture + alpha_offset)) / 255.0f;
    if (d_channels >= 3)
        color.z = (*(texture + blue_offset)) / 255.0f;
    if (d_channels >= 2)
        color.y = (*(texture + green_offset)) / 255.0f;
    if (d_channels >= 1)
        color.x = ( *(texture + red_offset)   ) / 255.0f;

    return color;
}

__device__ float lum(float4 color) {
    return (0.299 * color.x + 0.587 * color.y + 0.114 * color.z);
}

__global__ void render() {
    int2 coord = get_pixel_coord();
    
    float4 finalColor = make_float4(0, 0, 0, 0);
    float weight = 0;
    int size = 2;
    for (int i = -size; i <= size; i++)
    {
        for (int j = -size; j <= size; j++)
        {
            int2 c = coord + make_int2(i, j);
            if (c.x < 0 || c.x >= d_texture_size.x || c.y < 0 || c.y >= d_texture_size.y) {
                continue;
            }

            float w = sqrt((1 - abs(i) / (float)size) * (1 - abs(j) / (float)size));
            weight += w;
            finalColor += get_pixel(d_render_texture, c) * w;
        }
    }
    if (weight != 0) {
        finalColor /= (float)weight;
        finalColor *= 2.0;
        set_pixel(d_render_texture, finalColor, coord);
    }
    else {
        finalColor += get_pixel(d_render_texture, coord);
        set_pixel(d_render_texture, finalColor, coord);
    }
}

void clear_rt() {
    hipMemset(h_render_texture, 0, texture_mem_size);
}

void render_to_rt(uint8_t* pixels, bool blur) {
    if (blur) {
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_render_texture), &h_render_texture, sizeof(void*)));
        dim3 threads_per_block = dim3(8, 8, 1);
        dim3 blocks_per_grid = dim3((float)texture_size.x / threads_per_block.x, (float)texture_size.y / threads_per_block.y, 1);
        render << < blocks_per_grid, threads_per_block >> > ();
    }
    CUDA_CHECK(hipMemcpy(pixels, h_render_texture, texture_mem_size, hipMemcpyDeviceToHost));
}

int init_cuda_renderer(int render_width, int render_height, int channels) {
    texture_mem_size = render_height * render_width * sizeof(uint8_t) * MAX_CHANNELS;
    texture_size = make_int2(render_width, render_height);

    CUDA_CHECK(hipMalloc(&h_render_texture, texture_mem_size));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_texture_size), &texture_size, sizeof(float2), 0, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_channels), &channels, sizeof(int), 0, hipMemcpyHostToDevice));
	return 0;
}


////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////


__constant__ constexpr float GRAV_CONST = 1.0f;
__constant__ constexpr float DIST_SCALE = 0.5f;
__constant__ constexpr float MASS_SCALE = 1.0f;
Body* h_Bodies;
int body_count;

__global__ void calculate_nbody(Body* bodies, Body_Info* bodies_ext, int lowBound, int upBound, int count, float delta) {

    int midx = threadIdx.x + blockDim.x * blockIdx.x;
    if (midx >= count) {
        return;
    }

    Body me = bodies[midx];
    Body_Info me_ext = bodies_ext[midx];

    float2 curVel = me_ext.velocity;
    float2 total_force;

    for (size_t i = lowBound; i < upBound; i++)
    {
        if (i == midx) {
            continue;
        }

        Body t = bodies[i];
        float2 dir = t.position - me.position;
        float dist = length(dir) * DIST_SCALE;
        if (dist == 0.0) {
            continue;
        }


        if (dist <= 1.0) {
            float2 force = GRAV_CONST * me.mass * MASS_SCALE * t.mass * MASS_SCALE * normalize(dir) * powf(dist, 1.0f);
            total_force += force;
            curVel += force * delta / abs(me.mass * MASS_SCALE);
        }
        else {
            float dist2 = dist * dist;
            float2 force = GRAV_CONST * me.mass * MASS_SCALE * t.mass * MASS_SCALE * normalize(dir) / dist2;
            total_force += force;
            curVel += force * delta / abs(me.mass * MASS_SCALE);
        }
    }
    me.position += curVel * delta;

    //needed to avoid flickering
    float total_accel_length = length(total_force) / abs(me.mass * MASS_SCALE);
    float f0 = 1 / (1 + total_accel_length);
    float f1 = 1 / (1 + me_ext.previous_accel);
    if (f0 != 0 || f1 != 0) {
        me_ext.accel = max((total_accel_length * f0 + me_ext.previous_accel * f1) / (f0 + f1), 0.0f);
    }
    me_ext.previous_accel = max(total_accel_length, 0.0f);
    me_ext.velocity = curVel;

    bodies[midx] = me;
    bodies_ext[midx] = me_ext;
}

__device__ float4 get_color_from_value(float value, float scale) {

    float4 red = make_float4(1, 0, 0, 0);
    float4 yellow = make_float4(1, 1, 0, 0);
    float4 green = make_float4(0, 1, 0, 0);
    float4 cyan = make_float4(0, 1, 1, 0);
    float4 blue = make_float4(0, 0, 1, 0);
    float4 black = make_float4(0, 0, 0.5, 0);

    float4 finalCol = red;

    float scaler = scale;
    float inv_ten = 1.0 / (10.0f * scaler);

    if (value < 10 * scaler) {
        finalCol = (lerp(black, blue, value * inv_ten));
    }
    else if (value < 20 * scaler) {
        finalCol = (lerp(blue, cyan, (value - 10.0f * scaler) * inv_ten));
    }
    else if (value < 30 * scaler) {
        finalCol = (lerp(cyan, green, (value - 20.0f * scaler) * inv_ten));
    }
    else if (value < 40 * scaler) {
        finalCol = (lerp(green, yellow, (value - 30.0f * scaler) * inv_ten));
    }
    else if (value < 50 * scaler) {
        finalCol = (lerp(yellow, red, (value - 40.0f * scaler) * inv_ten));
    }
    else {
        finalCol = red;
    }

    return finalCol;
}

__global__ void draw_bodies(Body* bodies, Body_Info* bodies_ext, int count, uint8_t* outTexture, float colorScale, float renderScale, float2 renderOffset) {

    int midx = threadIdx.x + blockDim.x * blockIdx.x;
    if (midx >= count) {
        return;
    }
    Body me = bodies[midx];
    Body_Info me_ext = bodies_ext[midx];

    float2 mid_f = make_float2(d_texture_size.x, d_texture_size.y) * 0.5;
    float2 coord_f = make_float2(me.position.x + renderOffset.x, me.position.y + renderOffset.y);

    coord_f -= mid_f;
    coord_f *= renderScale;
    coord_f += mid_f;

    int2 coord = make_int2(floor(coord_f.x), floor(coord_f.y));
    if (coord.x > 0 && coord.x < d_texture_size.x
        && coord.y > 0 && coord.y < d_texture_size.y) {
        float val = me_ext.accel;
        float4 color = get_color_from_value(val, colorScale);
        set_pixel(outTexture, color, coord);
    }
}

float2 get_random_direction() {
    float x;
    float y;
    float2 dir;

    do
    {
        x = frandom(-1, 1);
        y = frandom(-1, 1);
        dir = make_float2(x, y);
    } while (length(dir) > 1.0);

    return dir;
}

Body* init_nbody(int count, int seed, float minMass, float maxMass, float2 spaceLow, float2 spaceHigh, bool spiral) {
    int memSize = sizeof(Body) * count;
    int memSize_ext = sizeof(Body_Info) * count;
    h_Bodies = (Body*)malloc(memSize);
    auto h_Bodies_ext = (Body_Info*)malloc(memSize_ext);
    memset(h_Bodies, 0, memSize);
    memset(h_Bodies_ext, 0, memSize_ext);
    srand(seed);

    float2 midPoint = (spaceHigh + spaceLow) * 0.5f;
    float extentLength = length(spaceHigh - midPoint) * 0.7;
    float spiral_center_mass = maxMass * 1000000;
    int neg_count = 0;
    int black_hole_count = 0;

    uint32_t total_system_mass = 0;

    for (size_t i = 0; i < count; i++)
    {

        h_Bodies[i].mass = frandom(minMass, maxMass);

        if (!spiral) {
            float randx = frandom(spaceLow.x, spaceHigh.x);
            float randy = frandom(spaceLow.y, spaceHigh.y);
            h_Bodies[i].position = make_float2(randx, randy);
        }
        else {
            h_Bodies[i].position = midPoint + get_random_direction() * extentLength;
        }

        h_Bodies_ext[i].velocity.x = 0;
        h_Bodies_ext[i].velocity.y = 0;

        if (neg_count == 63) {
            h_Bodies[i].mass *= -63.0f;
            neg_count = 0;
        }
        else {
            neg_count++;
        }

        if (spiral) {
            if (black_hole_count == count - 1) {
                h_Bodies[i].mass = frandom(spiral_center_mass * .1, spiral_center_mass * 0.25);
                black_hole_count = 0;
            }
            else {
                black_hole_count++;
            }
        }
        else {
            if (black_hole_count == ceilf((float)count * 0.25)) {
                h_Bodies[i].mass = maxMass * frandom(1000, 10000);
                black_hole_count = 0;
            }
            else {
                black_hole_count++;
            }
        }


        total_system_mass += h_Bodies[i].mass > 0.0 ? h_Bodies[i].mass : 0.0;
    }

    if (spiral) {
        for (size_t i = 0; i < count; i++)
        {
            float2 dir2mid = midPoint - h_Bodies[i].position;
            float dist = length(dir2mid) * DIST_SCALE;
            float3 dir2mid3 = normalize(make_float3(dir2mid.x, dir2mid.y, 0));
            float3 cross_dir = make_float3(0, 0, 1);
            float3 velocity_dir = normalize(cross(dir2mid3, cross_dir));
            float orbit_speed = sqrt(GRAV_CONST * (total_system_mass + spiral_center_mass / DIST_SCALE) / dist);
            float2 vel2 = make_float2(velocity_dir.x, velocity_dir.y) * orbit_speed;
            h_Bodies_ext[i].velocity = vel2;
        }
    }

    if (spiral) {
        h_Bodies[0].mass = spiral_center_mass;
        h_Bodies[0].position = midPoint;
        h_Bodies_ext[0].velocity *= 0;
    }

    body_count = count;
    CUDA_CHECK(hipMalloc(&d_Bodies, memSize));
    CUDA_CHECK(hipMalloc(&d_Bodies_ext, memSize_ext));
    CUDA_CHECK(hipMemset(d_Bodies, 0, memSize));
    CUDA_CHECK(hipMemset(d_Bodies_ext, 0, memSize_ext));

    CUDA_CHECK(hipMemcpy(d_Bodies, h_Bodies, memSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Bodies_ext, h_Bodies_ext, memSize, hipMemcpyHostToDevice));

    CUDA_CHECK(hipDeviceSynchronize());
    free(h_Bodies_ext);
    return h_Bodies;
}

void tick(float deltaTime, float colorScale, float renderScale, float2 renderOffset) {
    dim3 threads = dim3(64, 1, 1);
    dim3 blocks = dim3((uint)ceilf((float)body_count / threads.x), 1, 1);

    calculate_nbody << <blocks, threads >> > (d_Bodies, d_Bodies_ext, 0, body_count, body_count, deltaTime);
    CUDA_CHECK(hipDeviceSynchronize());

    draw_bodies << <blocks, threads >> > (d_Bodies, d_Bodies_ext, body_count, h_render_texture, colorScale, renderScale, renderOffset);
    CUDA_CHECK(hipDeviceSynchronize());
}